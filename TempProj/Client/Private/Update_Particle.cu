#include "hip/hip_runtime.h"
//#include "Update_Particle.cuh"


__device__  PARTICLE_DESC Play_TestInternal(const _uint& i, _float fTimeDelta, _matrix BoneMatrix, PARTICLE_DESC* input)
{
	PARTICLE_DESC desc = *input;
	desc.fCurrentLifeTime = 999.f;

	return desc;
}


__global__ void kernel_UpdateParticle(_float fTimeDelta, _matrix BoneMatrix, PARTICLE_DESC* input, PARTICLE_DESC* output, _int size)
{
	_int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < size)
	{
		*output = Play_TestInternal(i, fTimeDelta, BoneMatrix, input);
	}


}

__global__ void Kernel(int size)
{

}

__host__ void CudaMain_UpdateParticle(float fTimeDelta, _matrix BoneMatrix, PARTICLE_DESC* input, PARTICLE_DESC* output, _int size)
{
	
}

__host__ void CudaMain_UpdateParticleTEST(float fTimeDelta)
{
	//PARTICLE_DESC* pInputParticleDescToCuda;
	//PARTICLE_DESC* pOutputParticleDescToCuda;

	////_size_t Size = sizeof(PARTICLE_DESC) * m_tEffectParticleDesc.iMaxInstance;

	//hipMalloc((void**)&pInputParticleDescToCuda, 100);
	//hipMalloc((void**)&pOutputParticleDescToCuda, 100);

	////hipMemcpy(pInputParticleDescToCuda, &m_tParticleDescs[0], Size, hipMemcpyHostToDevice);

	//hipFree(pInputParticleDescToCuda);
	//hipFree(pOutputParticleDescToCuda);
}

